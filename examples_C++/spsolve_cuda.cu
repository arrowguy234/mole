/*
This isolated example illustrates how to solve a linear system where 'A' 
is of type sp_mat and 'b' is of type vec (Armadillo's types) using sparse QR 
factorization provided by cuSOLVER.

cuSOLVER doesn't provide a routine for sparse LU factorization on the GPU! The 
best alternative is QR factorization (based on the qualities of the mimetic Laplacian) 
even though it is twice as expensive.

Compile:
nvcc spsolve_cuda.cu -o spsolve_cuda -O3 -I./armadillo-10.2.1/include -lcusparse -lcusolver
*/

#include <hipsolver.h>
#define ARMA_ALLOW_FAKE_GCC
#include <armadillo>

using namespace arma;

double* cuda_qr(const double* values,
        const long long unsigned int a_nnz,
        const long long unsigned int rows,
        const long long unsigned int cols,
        const long long unsigned int* row_ind,
        const long long unsigned int* col_ptrs,
        double* b,
        double* x) {

  int nnz = (int)a_nnz;
  int* h_csccol_pts;
  int* h_cscRowInd;
  double* h_cscVal;

  h_csccol_pts = (int*)malloc(sizeof(int) * (cols + 1));
  for (int i = 0; i < cols + 1; ++i)
    h_csccol_pts[i] = (int)col_ptrs[i];

  h_cscRowInd = (int*)malloc(sizeof(int) * nnz);
  for (int i = 0; i < nnz; ++i)
    h_cscRowInd[i] = (int)row_ind[i];

  h_cscVal = (double*)malloc(sizeof(double) * nnz);
  memcpy(h_cscVal, values, sizeof(double) * nnz);

  size_t h_buffer = 0;
  double* d_buffer = nullptr;

  int* d_csccol_pts = nullptr;
  int* d_cscRowInd = nullptr;
  int* d_csrRowPtr = nullptr;
  int* d_csrColInd = nullptr;
  double* d_csrvalues = nullptr;
  double* d_cscVal = nullptr;

  hipsparseHandle_t sphandle = nullptr;
  hipsparseCreate(&sphandle);

  hipMalloc((void**)&d_cscVal, sizeof(double) * nnz);
  hipMalloc((void**)&d_csccol_pts, sizeof(int) * (cols + 1));
  hipMalloc((void**)&d_cscRowInd, sizeof(int) * nnz);
  hipMalloc((void**)&d_csrvalues, sizeof(double) * nnz);
  hipMalloc((void**)&d_csrRowPtr, sizeof(int) * (rows + 1));
  hipMalloc((void**)&d_csrColInd, sizeof(int) * nnz);

  hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseAction_t copyValues = HIPSPARSE_ACTION_NUMERIC;
  hipsparseCsr2CscAlg_t alg = HIPSPARSE_CSR2CSC_ALG2;
  hipDataType valType = HIP_R_64F;

  hipMemcpy(d_cscVal, h_cscVal, sizeof(double) * nnz, hipMemcpyHostToDevice);
  hipMemcpy(d_csccol_pts, h_csccol_pts, sizeof(int) * (cols + 1), hipMemcpyHostToDevice);
  hipMemcpy(d_cscRowInd, h_cscRowInd, sizeof(int) * nnz, hipMemcpyHostToDevice);

  hipsparseMatDescr_t descrA = nullptr;
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

  hipsparseCsr2cscEx2_bufferSize(sphandle,
                                cols,
                                rows,
                                nnz,
                                d_cscVal,
                                d_csccol_pts,
                                d_cscRowInd,
                                d_csrvalues,
                                d_csrRowPtr,
                                d_csrColInd,
                                valType,
                                copyValues,
                                idxBase,
                                alg,
                                &h_buffer);

  hipMalloc((void**)&d_buffer, sizeof(double) * h_buffer);

  hipsparseCsr2cscEx2(sphandle,
                     cols,
                     rows,
                     nnz,
                     d_cscVal,
                     d_csccol_pts,
                     d_cscRowInd,
                     d_csrvalues,
                     d_csrRowPtr,
                     d_csrColInd,
                     valType,
                     copyValues,
                     idxBase,
                     alg,
                     d_buffer);

  hipsolverSpHandle_t solhandle = nullptr;
  hipsolverSpCreate(&solhandle);

  double* d_b = nullptr;
  hipMalloc((void**)&d_b, sizeof(double) * rows);

  double tol = 1e-12;
  const int reorder = 1;
  int singularity = 0;

  double* d_x = nullptr;
  hipMalloc((void**)&d_x, sizeof(double) * cols);

  hipMemcpy(d_b, b, sizeof(double) * rows, hipMemcpyHostToDevice);

  hipsolverSpDcsrlsvqr(solhandle,
                      cols,
                      nnz,
                      descrA,
                      d_csrvalues,
                      d_csrRowPtr,
                      d_csrColInd,
                      d_b,
                      tol,
                      reorder,
                      d_x,
                      &singularity);

  if (0 <= singularity)
        cout << "Matrix is singular\n";

  x = (double*)malloc(sizeof(double) * cols);

  hipMemcpy(x, d_x, sizeof(double) * cols, hipMemcpyDeviceToHost);

  hipsolverSpDestroy(solhandle);
  hipsparseDestroy(sphandle);
  hipsparseDestroyMatDescr(descrA);
  free(h_cscVal);
  free(h_csccol_pts);
  free(h_cscRowInd);
  hipFree(d_cscVal);
  hipFree(d_csccol_pts);
  hipFree(d_cscRowInd);
  hipFree(d_csrvalues);
  hipFree(d_csrRowPtr);
  hipFree(d_csrColInd);
  hipFree(d_buffer);
  hipFree(d_x);
  hipFree(d_b);

  return x;
}

vec spsolve_cuda(sp_mat A, vec b) {
  const double* values = A.values;
  const long long unsigned int nnz = A.n_nonzero;
  const long long unsigned int rows = A.n_rows;
  const long long unsigned int cols = A.n_cols;
  const long long unsigned int* row_indices = A.row_indices;
  const long long unsigned int* col_ptrs = A.col_ptrs;

  double* b_ = b.memptr();
  double* x = nullptr;

  return vec(cuda_qr(values, nnz, rows, cols, row_indices, col_ptrs, b_, x), A.n_cols);
}

int main() {
  int size = 1000;
  sp_mat A = sprandn<sp_mat>(size, size, 0.05); // 5% density
  vec b(size, fill::randu);

  vec x = spsolve_cuda(A, b);

  return 0;
}
